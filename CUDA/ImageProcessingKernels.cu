//Kernel code:
extern "C"  
{   
	
#include <hip/hip_runtime.h>
#include <math.h>

    // Device code
    __global__ void ConvertRgb24ToGray8(char* source, char* result, double redPortion, double greenPortion, double bluePortion, int N)
    {
        int indexResult = blockDim.x * blockIdx.x + threadIdx.x;
		
		if (indexResult < N)
		{
			int indexSource = indexResult * 3;
			
			double value = redPortion * source[indexSource] + greenPortion * source[indexSource + 1] + bluePortion * source[indexSource + 2];
		
			if (value >= 0xFF)
				result[indexResult] = 0xFF;
			else if (value <= 0x00)
				result[indexResult] = 0x00;
			else
				result[indexResult] = (char)value;
		}
    }
	
	__global__ void ConvertGray8ToRgb24(char* source, char* result, int N)
    {
        int indexSource = blockDim.x * blockIdx.x + threadIdx.x;
		
		if (indexSource < N)
		{
			int indexResult = indexSource * 3;
		
			result[indexResult++] = source[indexSource];
			result[indexResult++] = source[indexSource];
			result[indexResult]   = source[indexSource];
		}
    }
	
	__global__ void Invert(char* source, char* result, int N)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        
		if (i < N)
            result[i] = ~source[i];
    }
	
	__global__ void ApplyThreshold(char* source, char* result, int threshold, int N)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        
		if (i < N)
		{
			if (source[i] < threshold)
			{
				result[i] = 0x00;
			}
			else
			{
				result[i] = 0xFF;
			}
		}
    }
	
	__global__ void AdjustBrightness(char* source, char* result, int value, int N)
    {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        
		if (i < N)
		{
			int newValue = source[i] + value;
		
			if (newValue < 0x00)
				result[i] = 0x00;
			else if (newValue > 0xFF)
				result[i] = 0xFF;
			else
				result[i] = (char)newValue;
		}
    }
	
	__global__ void AdjustContrast(char* source, char* result, int factor, int N)
    {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        
		if (i < N)
		{
			float newValue = factor * (source[i] - 128.0) + 128.0;

			if (newValue > 0xFF)
				result[i] = 0xFF;
			else if (newValue < 0x00)
				result[i] = 0x00;
			else
				result[i] = (char)round(newValue);
		}
    }
	
	__global__ void ApplyU8LookupTablePerChannel(char* source, char* result, char* lookupTable, int N)
    {
		__shared__ char lut[256];

		if (threadIdx.x < 256)
			lut[threadIdx.x] = lookupTable[threadIdx.x];
	
		__syncthreads();
	
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        
		if (i < N)
		{
			result[i] = lut[source[i]];
		}
    }
}